/* util.cu: CUDA convenience functions. */

#include "roughsort.h"

// TODO: propagate __FILE__ and __LINE__ from caller if GDB proves inadequate
#define CHECK(r) cuCheck(r, __FILE__, __LINE__)

inline void cuCheck(hipError_t r, const char* fname, const size_t lnum) {
  if (r != hipSuccess) {
    fatal("CUDA error at line %d in %s\n", lnum, fname);
  }
}

size_t cuMemAvail() {
  size_t free, total;
  CHECK(hipMemGetInfo(&free, &total));
  return free;
}

void* cuMalloc(size_t size) {
  void* p;
  CHECK(hipMalloc(&p, size));
  return p;
}

void cuFree(void* p) {
  CHECK(hipFree(p));
}

void cuClear(void* p, size_t size) {
  CHECK(hipMemset(p, 0, size));
}

void cuUpload(void* devDst, const void* hostSrc, size_t size) {
  CHECK(hipMemcpy(devDst, hostSrc, size, hipMemcpyHostToDevice));
}

void cuDownload(void* hostDst, const void* devSrc, size_t size) {
  CHECK(hipMemcpy(hostDst, devSrc, size, hipMemcpyDeviceToHost));
}

void cuPin(void* p, size_t size) {
  CHECK(hipHostRegister(p, size, hipHostRegisterPortable));
}

void cuUnpin(void* p) {
  CHECK(hipHostUnregister(p));
}
