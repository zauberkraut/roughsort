#include "hip/hip_runtime.h"
/* sort.cu: CUDA kernels for parallel sorts on an Nvidia GPU. */

#include "thrust/device_ptr.h"
#include "thrust/sort.h"
#include <thrust/extrema.h>
#include "roughsort.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include "math_functions.h"
#include "sm_20_atomic_functions.h"
#include <iostream>


using namespace std;

#define CHECK(r) cuCheck(r, __FILE__, __LINE__) //TODO: get util.h created for windows branch
inline void cuCheck(hipError_t r, const char* fname, const size_t lnum);
__device__ void devCheckIfSorted(int32_t* a, int n, int local_id, bool * sorted);



void devMergesort(int32_t* const a, const int n) {
  thrust::device_ptr<int32_t> devA(a);
  thrust::stable_sort(devA, devA + n);
}

void devQuicksort(int32_t* const a, const int n) {
  thrust::device_ptr<int32_t> devA(a);
  thrust::sort(devA, devA + n);
}

void devRoughsort(int32_t* const a, const int n) 
{
  thrust::device_ptr<int32_t> devA(a);
  // TODO
}


__global__ void devCheckSortednessCallee(int32_t* const a, const int n, int * k, int * b, int * c, int * d, int * r, bool * sorted, int tpbBits, int g0Bits, int g1Bits)
{

	unsigned long long threadXBits = (unsigned long long)threadIdx.x;
	unsigned long long gridXBits = (unsigned long long)(blockIdx.x) << tpbBits;
	unsigned long long gridYBits = (unsigned long long)(blockIdx.y) << tpbBits;
	gridYBits = gridYBits << g0Bits; //arch specific, need to pass the max values
	unsigned long long thread_id = gridXBits | gridYBits | threadXBits;

	int local_id = -1;
	if (thread_id < n)
		local_id = (int)thread_id;
	else
		return;

	if(local_id == 0)
		atomicAdd(r, tpbBits + g0Bits);

	b[local_id] = a[local_id];
	c[local_id] = a[local_id];
	d[local_id] = 0;
	//max-prefix - 
	for (int r = 0; r <= log2((float)(n)); r++)
	{
		*sorted = true;
		devCheckIfSorted(b, n, local_id, sorted);
		if (*sorted == true)
		{
			break;
		}
		else if (local_id - exp2((float)r) >= 0)
		{
			int idx = local_id - exp2((float)r);
			b[local_id] = max(b[local_id], b[idx]);
		}
	}
	//min-prefix
	for (int r = 0; r <= log2((float)(n)); r++)
	{
		*sorted = true;
		devCheckIfSorted(c, n, local_id, sorted);
		if (*sorted == true)
		{
			break;
		}
		else if (local_id + exp2((float)r) < n)
		{
			int idx = local_id + exp2((float)r);
			c[local_id] = min(c[local_id], c[idx]);
		}
	}
	
	__syncthreads();

	int i = local_id;
	for (int j = n - 1; j >= 0; j--) {
		if (j <= i && i >= 0 && c[i] <= b[j] &&
			(j == 0 || c[i] >= b[j - 1])) {
			d[i] = i - j;
		}
	}

	//use thrust maxelement to find max of d, which is k
	
	*k = *(thrust::max_element(thrust::device, &d[0], &d[n]));
	

}

__device__ void devCheckIfSorted(int32_t* a, int n, int local_id, bool * sorted)
{

	if (a[local_id] > a[max(local_id + 1, n - 1)])
		*sorted = false;

}

void devCheckSortedness(int32_t* const a, const int n)
{
	
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);


	unsigned long long max = n;
	unsigned long long threadblockX = max / deviceProp.maxThreadsPerBlock > 1 ? deviceProp.maxThreadsPerBlock : max;
	threadblockX = threadblockX == 0 ? 1 : threadblockX;
	std::cout << "Thread block X: " << threadblockX << std::endl;
	std::cout << "Max block X: " << deviceProp.maxThreadsPerBlock << std::endl;

	unsigned long long threadblockY = 1;
	std::cout << "Thread block Y: " << threadblockY << std::endl;

	unsigned long long threadblockZ = 1;
	std::cout << "Thread block Z: " << threadblockZ << std::endl;

	//calculates required grid X dimension based on the dimension available on device
	unsigned long long gridX = max / (deviceProp.maxGridSize[0]) / threadblockX / threadblockY / threadblockZ > 1 ? deviceProp.maxGridSize[0] : max / threadblockX / threadblockY / threadblockZ + (max % (threadblockX * threadblockY * threadblockZ) > 0 ? 1 : 0);
	gridX = gridX == 0 ? 1 : gridX;
	std::cout << "Grid X: " << gridX << std::endl;
	std::cout << "Max Grid X: " << deviceProp.maxGridSize[0] << std::endl;

	//calculates required grid Y dimension based on the dimension available on device
	unsigned long long gridY = max / threadblockX / threadblockY / threadblockZ / gridX / deviceProp.maxGridSize[1] > 1 ? deviceProp.maxGridSize[1] : max / threadblockX / threadblockY / threadblockZ / gridX + (max % (threadblockX * threadblockY * threadblockZ * gridX) > 0 ? 1 : 0);
	gridY = gridY == 0 ? 1 : gridY;
	std::cout << "Grid Y: " << gridY << std::endl;
	std::cout << "Max Grid Y: " << deviceProp.maxGridSize[1] << std::endl;

	dim3 dimBlock(threadblockX, threadblockY, threadblockZ);
	dim3 dimGrid(gridX, gridY, 1);

	int * b = (int*)cuMalloc(sizeof(int) * n);
	int * c = (int*)cuMalloc(sizeof(int) * n);
	int * d = (int*)cuMalloc(sizeof(int) * n);
	int * r = (int*)cuMalloc(sizeof(int));
	int * k = (int*)cuMalloc(sizeof(int));
	bool * sorted = (bool*)cuMalloc(sizeof(bool));
	hipDeviceProp_t * devProp = (hipDeviceProp_t *)cuMalloc(sizeof(hipDeviceProp_t));

	int r_host = 0;
	int sorted_host = false;

	hipMemcpy(r, &r_host, 1, hipMemcpyHostToDevice);
	hipMemcpy(sorted, &sorted_host, 1, hipMemcpyHostToDevice);
	hipMemcpy(devProp, &deviceProp, 1, hipMemcpyHostToDevice);
	devCheckSortednessCallee << <dimGrid, dimBlock >> >(a, n, k, b, c, d, r, sorted, log2(deviceProp.maxThreadsPerBlock), log2(deviceProp.maxGridSize[0]), log2(deviceProp.maxGridSize[1]));

	hipDeviceSynchronize();
	hipDeviceSynchronize();

	int k_host;
	int * b_host = (int*)malloc(sizeof(int)* n);
	int * c_host = (int*)malloc(sizeof(int)* n);
	int * d_host = (int*)malloc(sizeof(int)* n);
	hipMemcpy(&k_host, k, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(b_host, b, sizeof(int) * n, hipMemcpyDeviceToHost);
	hipMemcpy(c_host, c, sizeof(int) * n, hipMemcpyDeviceToHost);
	hipMemcpy(d_host, d, sizeof(int) * n, hipMemcpyDeviceToHost);
	hipMemcpy(&r_host, r, sizeof(int), hipMemcpyDeviceToHost);


	CHECK(hipGetLastError());
	std::cout << "K value: " << k_host << std::endl;
	std::cout << "R value: " << r_host << std::endl;
	
	if(n<=256)
	for (int i = 0; i < n; i++)
	{
		cout << b_host[i] << "\t" << c_host[i] << "\t" << d_host[i] << endl;
	}
	
	cuFree(b);
	CHECK(hipGetLastError());
	cuFree(c);
	CHECK(hipGetLastError());
	cuFree(d);
	CHECK(hipGetLastError());
	cuFree(r);
	CHECK(hipGetLastError());
	cuFree(k);
	CHECK(hipGetLastError());


}


