/* cuda_sort.cu: CUDA kernels for parallel sorts on an Nvidia GPU. */


#include <hip/hip_runtime.h>
#include <cassert>

// Helper CUDA functions...

size_t cuMemAvail() {
  size_t free, total;
  assert(hipSuccess == hipMemGetInfo(&free, &total));
  return free;
}

void* cuMalloc(size_t size) {
  void* p;
  assert(hipSuccess == hipMalloc(&p, size));
  return p;
}

void cuFree(void* p) { assert(hipSuccess == hipFree(p)); }

void cuClear(void* p, size_t size) {
  assert(hipSuccess == hipMemset(p, 0, size));
}

void cuUpload(void* devDst, const void* hostSrc, size_t size) {
  assert(hipSuccess == hipMemcpy(devDst, hostSrc, size,
                                   hipMemcpyHostToDevice));
}

void cuDownload(void* hostDst, const void* devSrc, size_t size) {
  assert(hipSuccess == hipMemcpy(hostDst, devSrc, size,
         hipMemcpyDeviceToHost));
}

void cuPin(void* p, size_t size) {
  assert(hipSuccess == hipHostRegister(p, size, hipHostRegisterPortable));
}

void cuUnpin(void* p) {
  assert(hipSuccess == hipHostUnregister(p));
}

// Kernels...

__global__ void kernSquare(int* a, const int n) {
  for (int i = 0; i < n; i++) {
    a[i] *= a[i];
  }
}

void cuSquare(int* a, int n) {
  kernSquare<<<1, 1>>>(a, n);
}
